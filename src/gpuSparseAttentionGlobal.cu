#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <chrono>

#include "gpuSparseAttentionGlobal.h"



void gpuSparseAttentionGlobal(int N, int D_MODEL, int N_HEAD) {
    int d_k = D_MODEL / N_HEAD;
    float sqrt_d_k = sqrt(d_k);

    float *query, *key, *value, *attn_scores, *result;
    hipMalloc((void **)&query, N_HEAD*N*d_k * sizeof(float));
    hipMalloc((void **)&key, N_HEAD*N*d_k * sizeof(float));
    hipMalloc((void **)&value, N_HEAD*N*d_k * sizeof(float));
    hipMalloc((void **)&attn_scores, N_HEAD * N*N * sizeof(float));
    hipMalloc((void **)&result, N_HEAD*N*d_k * sizeof(float));

    // =============================================================================================
    // Tiling + random sparse

    hipEvent_t start, stop;
    float elapsedTime = 0.0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);

    // Kernel Here

    hipDeviceSynchronize();
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    printf("gpu global attention (tile parallelization using shared mem): %f microseconds\n\n" ,elapsedTime*1000);

}
