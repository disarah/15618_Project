#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <chrono>

#include "gpuSparseAttention.h"


__global__ void sparseAttention(float* query, float* key, float* value,
                                int N, int N_HEAD, int d_k, float sqrt_d_k,
                                float* attn_scores, float* result, int ws) {

    int h = threadIdx.x;     // local pixel x index in this block
    float score;

    // query x key^T
    for(int n1 = 0; n1 < N; n1++) {
        int start = int(n1 / ws) * ws;
        int end = start + ws;
        for(int n2 = start; n2 < end; n2++) {
            score = 0.0;
            for(int d=0; d < d_k; d++) {
                score += query[h*N*d_k + n1*d_k + d] * key[h*N*d_k + n2*d_k + d];
            }
            attn_scores[h*N*N + n1*N + n2] = score / sqrt_d_k;
        }
    }

    for(int n1 = 0; n1 < N; n1++) {
        float sum = 0.0;
        int start = int(n1 / ws) * ws;
        int end = start + ws;
        for(int n2 = start; n2 < end; n2++) {
            sum += attn_scores[h*N*N + n1*N + n2];
        }
        for(int n2 = start; n2 < end; n2++) {
            attn_scores[h*N*N + n1*N + n2] /= sum;
        }
    }

    for(int n1 = 0; n1 < N; n1++) {
        int start = int(n1 / ws) * ws;
        int end = start + ws;
        for(int d = 0; d < d_k; d++) {
            float sum = 0.0;
            for(int n2 = start; n2 < end; n2++) {
                sum += attn_scores[h*N*N + n1*N + n2] * value[h*N*d_k + n2*d_k + d];
            }
            result[h*N*d_k + n1*d_k + d] = sum;
        }
    }
}

void gpuSparseAttention(int N, int D_MODEL, int N_HEAD) {
    int d_k = D_MODEL / N_HEAD;
    float sqrt_d_k = sqrt(d_k);
    int ws = 64; // window size

    float *query, *key, *value, *attn_scores, *result;
    hipMalloc((void **)&query, N_HEAD*N*d_k * sizeof(float));
    hipMalloc((void **)&key, N_HEAD*N*d_k * sizeof(float));
    hipMalloc((void **)&value, N_HEAD*N*d_k * sizeof(float));
    hipMalloc((void **)&attn_scores, N_HEAD * N*N * sizeof(float));
    hipMalloc((void **)&result, N_HEAD*N*d_k * sizeof(float));


    dim3 threadPerBlock(N_HEAD);
    dim3 numBlock(1);

    hipEvent_t start, stop;
    float elapsedTime = 0.0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);
    // auto beg = std::chrono::high_resolution_clock::now();
	sparseAttention<<<numBlock, threadPerBlock>>>(query, key, value, N, N_HEAD, d_k, sqrt_d_k, attn_scores, result, ws);

    hipDeviceSynchronize();
    // auto end = std::chrono::high_resolution_clock::now();
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    printf("gpu sparse attention: %fms\n" ,elapsedTime);

}
